#include <fstream>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <math.h>

using namespace std;

//Execute 1 thread per pixel of output image.
//Requires no atomics
__global__ void encode_per_pixel_kernel(uchar4* const d_destImg,
                              const char* const d_binData,
                              int numBytesData)
{
  int pixel = threadIdx.x + blockDim.x * blockIdx.x;
  if(pixel >= numBytesData)
    return;
  
  //Pixel 5 is at byte 3.
  int dataStart = pixel / 2 + 1;
  int nibble = pixel % 2;

  char dataByte = d_binData[dataStart];
  
  //Can't do next part in a loop because we have to access differently (x,y,z,w)
  
  //Channel 0 (first bit in the nibble)
  int offset = (7 - 1 * nibble);
  char mask = 1 << offset;
  char bit = (dataByte & mask) >> offset;
  d_destImg[pixel].x += bit;
  
  //Channel 1 (2nd bit)
  offset -= 1;
  mask >>= 1;
  bit = (dataByte & mask) >> offset;
  d_destImg[pixel].y += bit;
  
  //Channel 2 (3rd bit)
  offset -= 1;
  mask >>= 1;
  bit = (dataByte & mask) >> offset;
  d_destImg[pixel].z += bit;
  
  //Channel 3 (4th bit)
  offset -= 1;
  mask >>= 1;
  bit = (dataByte & mask) >> offset;
  d_destImg[pixel].z += bit;
  
}

/**

| 10 11 12 15 ; 11 255 12 0 |
| 15 10 13 5  ; 15 14 19 80 | Original image (each set of 4 is 1 pixel).
| 12 14 16 21 ; 14 18 10 16 |
| 10 10 10 10 ; 10 10 10 10 |

+

[ 1001 0110 1111 0000 1010 0101 0100 1100]  Data file

= 

| 11 11 12 16 ; 11 0  13 0  |
| 15 11 14 6  ; 15 14 19 80 | Encoded image
| 13 14 16 21 ; 14 19 10 17 |
| 10 11 10 10 ; 11 11 10 10 |
 
 */
void encode_parallel(const uchar4* const h_sourceImg,
                     uchar4* const h_destImg,
                     const char* const h_binData,
                     int numBytesData,
                     const size_t numRowsSource, const size_t numColsSource)
{

  //Allocate device memory
  uchar4* d_destImg;
  char* d_binData;
  hipMalloc(&d_destImg, sizeof(uchar4) * numRowsSource * numColsSource);
  hipMalloc(&d_binData, sizeof(char) * numBytesData);
  
  hipMemcpy(d_destImg, h_sourceImg, sizeof(uchar4) * numRowsSource * numColsSource, hipMemcpyHostToDevice); 
  hipMemcpy(d_binData, h_binData, numBytesData, hipMemcpyHostToDevice);

  //Execute 1 thread per pixel of output image.
  //This means 1 thread per 4 bits of data.
  int numThreads = ceil(numBytesData / 4.0);
  int blockSize = 1024;
  int numBlocks = ceil((float)numThreads / blockSize);
  
  cout << "numBlocks: " << numBlocks << " blockSize: " << blockSize << " numThreads: " << numThreads << endl;
  
  encode_per_pixel_kernel<<<numBlocks, numThreads>>>(d_destImg, d_binData, numBytesData);
  
  hipMemcpy(h_destImg, d_destImg, sizeof(uchar4) * numRowsSource * numColsSource, hipMemcpyDeviceToHost);
  
  //Free memory
  hipFree(d_destImg);
  hipFree(d_binData);
                  
}
